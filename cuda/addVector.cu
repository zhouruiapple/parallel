
#include <hip/hip_runtime.h>
#include<stdio.h>
#include <time.h>
#define N 256

//
// Based on:
// http://www.nvidia.com/docs/IO/116711/sc11-cuda-c-basics.pdf
//

__global__ void addvec(int*a, int*b, int*c) { 

    if(threadIdx.x<N) 
        c[threadIdx.x]=a[threadIdx.x]+b[threadIdx.x]; 
} 

int main(void){

    clock_t begin, end;
    double time_spent;
    begin = clock();

    int a[N], b[N], c[N];       //  host variables
    int *dev_a, *dev_b, *dev_c; //  device variables
  

    //allocate memory on the GPU 
    hipMalloc((void**)&dev_a,N * sizeof(int)); 
    hipMalloc((void**)&dev_b,N * sizeof(int)); 
    hipMalloc((void**)&dev_c,N * sizeof(int)); 

    //Fill arrays with data
    for(int i=0; i<N;i++){ 
        a[i]=i; 
        b[i]=i+1; 
    }

    //copy arrays to GPU
    hipMemcpy(dev_a,a,N * sizeof(int),hipMemcpyHostToDevice); 
    hipMemcpy(dev_b,b,N * sizeof(int),hipMemcpyHostToDevice); 

    //START KERNEL
    addvec<<<2,N>>>(dev_a,dev_b,dev_c); 

    //Copy result array to host 
    hipMemcpy(c,dev_c,N * sizeof(int),hipMemcpyDeviceToHost); 

    //Print the result ( comment if you want to mesure time ) 
    /*for(int i=0; i<N; i++){ 
        printf("%d+%d=%d\n",a[i],b[i],c[i]); 
    } */

    // Free device memory
    hipFree(dev_a); 
    hipFree(dev_b); 
    hipFree(dev_c); 

    end = clock();
    time_spent = (double)(end - begin) / CLOCKS_PER_SEC;

    printf(" Time : %f\n", time_spent );

    return 0; 
} 
